/**********************************************************************/
// An unoptimized Naive N-Body solver for Gravity Simulations         //
// G is assumed to be 1.0                                             //
// Course Material for HPCSE-II, Spring 2019, ETH Zurich              //
// Authors: Sergio Martin                                             //
// License: Use if you like, but give us credit.                      //
/**********************************************************************/


#include <hip/hip_runtime.h>
#include "string.h"
#include <chrono>
#include <math.h>
#include <stdio.h>

#define BLOCKSIZE 1024

void checkCUDAError(const char *msg);

__global__ void forceKernel(double *xPos, double *yPos, double *zPos,
                            double *mass, double *xFor, double *yFor,
                            double *zFor, size_t N) {
  size_t mIdx = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ double x[BLOCKSIZE];
  __shared__ double y[BLOCKSIZE];
  __shared__ double z[BLOCKSIZE];
  __shared__ double m[BLOCKSIZE];

  double x_force = 0;
  double y_force = 0;
  double z_force = 0;

  double x_mIdx = xPos[mIdx];
  double y_mIdx = yPos[mIdx];
  double z_mIdx = zPos[mIdx];
  double m_mIdx = mass[mIdx];

  for (size_t b = 0; b < N; b += BLOCKSIZE) {
    x[threadIdx.x] = xPos[b + threadIdx.x];
    y[threadIdx.x] = yPos[b + threadIdx.x];
    z[threadIdx.x] = zPos[b + threadIdx.x];
    m[threadIdx.x] = mass[b + threadIdx.x];

    __syncthreads();

#pragma unroll 16
    for (size_t i = 0; i < BLOCKSIZE; ++i) {
      double xDist = x_mIdx - x[i];
      double yDist = y_mIdx - y[i];
      double zDist = z_mIdx - z[i];

      double r = rsqrt(xDist * xDist + yDist * yDist + zDist * zDist + 1e-16);
      double tmp = m_mIdx * m[i] * (r * r * r);

      x_force += xDist * tmp;
      y_force += yDist * tmp;
      z_force += zDist * tmp;
    }

    __syncthreads();
  }

  xFor[mIdx] = x_force;
  yFor[mIdx] = y_force;
  zFor[mIdx] = z_force;
}

int main(int argc, char *argv[]) {
  size_t N0 = 80;
  size_t N = N0 * N0 * N0;

  // Initializing N-Body Problem

  double *xPos = (double *)calloc(N, sizeof(double));
  double *yPos = (double *)calloc(N, sizeof(double));
  double *zPos = (double *)calloc(N, sizeof(double));
  double *xFor = (double *)calloc(N, sizeof(double));
  double *yFor = (double *)calloc(N, sizeof(double));
  double *zFor = (double *)calloc(N, sizeof(double));
  double *mass = (double *)calloc(N, sizeof(double));

  size_t current = 0;
  for (size_t i = 0; i < N0; i++)
    for (size_t j = 0; j < N0; j++)
      for (size_t k = 0; k < N0; k++) {
        xPos[current] = i;
        yPos[current] = j;
        zPos[current] = k;
        mass[current] = 1.0;
        xFor[current] = 0.0;
        yFor[current] = 0.0;
        zFor[current] = 0.0;
        current++;
      }

  // Allocating and initializing GPU memory

  double *dxPos;
  hipMalloc((void **)&dxPos, sizeof(double) * N);
  checkCUDAError("Unable to allocate storage on the device");
  double *dyPos;
  hipMalloc((void **)&dyPos, sizeof(double) * N);
  checkCUDAError("Unable to allocate storage on the device");
  double *dzPos;
  hipMalloc((void **)&dzPos, sizeof(double) * N);
  checkCUDAError("Unable to allocate storage on the device");
  double *dxFor;
  hipMalloc((void **)&dxFor, sizeof(double) * N);
  checkCUDAError("Unable to allocate storage on the device");
  double *dyFor;
  hipMalloc((void **)&dyFor, sizeof(double) * N);
  checkCUDAError("Unable to allocate storage on the device");
  double *dzFor;
  hipMalloc((void **)&dzFor, sizeof(double) * N);
  checkCUDAError("Unable to allocate storage on the device");
  double *dmass;
  hipMalloc((void **)&dmass, sizeof(double) * N);
  checkCUDAError("Unable to allocate storage on the device");

  hipMemcpy(dxPos, xPos, sizeof(double) * N, hipMemcpyHostToDevice);
  checkCUDAError("Failed Initial Conditions Memcpy");
  hipMemcpy(dyPos, yPos, sizeof(double) * N, hipMemcpyHostToDevice);
  checkCUDAError("Failed Initial Conditions Memcpy");
  hipMemcpy(dzPos, zPos, sizeof(double) * N, hipMemcpyHostToDevice);
  checkCUDAError("Failed Initial Conditions Memcpy");
  hipMemcpy(dxFor, xFor, sizeof(double) * N, hipMemcpyHostToDevice);
  checkCUDAError("Failed Initial Conditions Memcpy");
  hipMemcpy(dyFor, yFor, sizeof(double) * N, hipMemcpyHostToDevice);
  checkCUDAError("Failed Initial Conditions Memcpy");
  hipMemcpy(dzFor, zFor, sizeof(double) * N, hipMemcpyHostToDevice);
  checkCUDAError("Failed Initial Conditions Memcpy");
  hipMemcpy(dmass, mass, sizeof(double) * N, hipMemcpyHostToDevice);
  checkCUDAError("Failed Initial Conditions Memcpy");

  // Calculating Kernel Geometry
  size_t threadsPerBlock = BLOCKSIZE;
  size_t blocksPerGrid = ceil(double(((double)N) / ((double)threadsPerBlock)));

  // Running Force-calculation kernel
  auto startTime = std::chrono::system_clock::now();
  forceKernel<<<blocksPerGrid, threadsPerBlock>>>(dxPos, dyPos, dzPos, dmass,
                                                  dxFor, dyFor, dzFor, N);
  checkCUDAError("Failed Force Kernel");
  hipDeviceSynchronize();
  auto endTime = std::chrono::system_clock::now();

  hipMemcpy(xFor, dxFor, sizeof(double) * N, hipMemcpyDeviceToHost);
  checkCUDAError("Failed Final Conditions Memcpy");
  hipMemcpy(yFor, dyFor, sizeof(double) * N, hipMemcpyDeviceToHost);
  checkCUDAError("Failed Final Conditions Memcpy");
  hipMemcpy(zFor, dzFor, sizeof(double) * N, hipMemcpyDeviceToHost);
  checkCUDAError("Failed Final Conditions Memcpy");

  double netForce = 0.0;
  double absForce = 0.0;
  for (size_t i = 0; i < N; i++)
    netForce += xFor[i] + yFor[i] + zFor[i];
  for (size_t i = 0; i < N; i++)
    absForce += abs(xFor[i] + yFor[i] + zFor[i]);

  printf("     Net Force: %.12f\n", netForce);
  printf("Absolute Force: %.12f\n", absForce);

  if (isfinite(netForce) == false) {
    printf("Verification Failed: Net force is not a finite value!\n");
    exit(-1);
  }
  if (fabs(netForce) > 0.00001) {
    printf("Verification Failed: Force equilibrium not conserved!\n");
    exit(-1);
  }
  if (isfinite(absForce) == false) {
    printf("Verification Failed: Absolute Force is not a finite value!\n");
    exit(-1);
  }

  printf("Time: %.8fs\n",
         std::chrono::duration<double>(endTime - startTime).count());
  return 0;
}

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "CUDA Error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}
